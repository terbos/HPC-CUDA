#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define WIDTH 500
#define BLOCK_WIDTH 16

__global__ 
void matMulKernel(float *d_M, float *d_N, float *d_P, int Width)
{ 
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    int k;
    if ((Row<Width)&&(Col<Width)){
        float Pvalue = 0.0;
        for(k=0;k<Width;k++)
            Pvalue += d_M[Row*Width+k]*d_N[k*Width+Col];
        d_P[Row*Width+Col] = Pvalue;
    }
}

void matMulDevice(float *h_M, float *h_N, float *h_P, int Width)
{
    int size = Width * Width * sizeof(float); 
    float *d_M, *d_N, *d_P;
// Step 1: Allocate and Load M, N to device memory 
    hipMalloc((void **)&d_M, size);
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_N, size);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
// Step 2: Allocate P on the device
    hipMalloc((void **)&d_P, size);
// Step 3a: Set up execution configuration
   int numBlocks = ceil(Width/(float)BLOCK_WIDTH);
   dim3 dimGrid(numBlocks,numBlocks);
   dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
// Step 3b: Launch the device computation threads!
   matMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, Width);
// Step 4: Copy back result, and free memory on device
   hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
   hipFree(d_M); hipFree(d_N); hipFree(d_P);
}

int main()
{
    float *h_M, *h_N, *h_P;
    int i, n = WIDTH, size=sizeof(float)*n*n;
    h_P = (float *)malloc(size);
    h_M = (float *)malloc(size);
    h_N = (float *)malloc(size);
    for(i=0;i<n*n;i++){*(h_M+i)=(float)i; 
                     *(h_N+i)=(float)i;}
    matMulDevice(h_M,h_N,h_P,n);
}
