#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define NSTEPS 500
#define TX 16
#define TY 32
#define NPTSX 200
#define NPTSY 200

__global__ 
void performUpdatesKernel(float *d_phi, float *d_oldphi, int *d_mask, int nptsx, int nptsy)
{
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    int x = Row*nptsx+Col;
    int xm = x-nptsx;
    int xp = x+nptsx;

    if(Col<nptsx && Row<nptsy)
        if (d_mask[x]) d_phi[x] = 0.25f*(d_oldphi[x+1]+d_oldphi[x-1]+d_oldphi[xp]+d_oldphi[xm]);
}
__global__
void doCopyKernel(float *d_phi, float *d_oldphi, int *d_mask, int nptsx, int nptsy)
{
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    int x = Row*nptsx+Col;

    if(Col<nptsx && Row<nptsy)
        if (d_mask[x]) d_oldphi[x] = d_phi[x];
}

void performUpdates(float *h_phi, float * h_oldphi, int *h_mask, int nptsx, int nptsy, int nsteps)
{
    float *d_phi, *d_oldphi;
    int *d_mask;
    int k;
    int sizef = sizeof(float)*nptsx*nptsy;
    int sizei = sizeof(int)*nptsx*nptsy;
    hipMalloc((void **)&d_phi,sizef);
    hipMalloc((void **)&d_oldphi,sizef);
    hipMalloc((void **)&d_mask,sizei);
    hipMemcpy(d_oldphi,h_oldphi,sizef,hipMemcpyHostToDevice);
    hipMemcpy(d_mask,h_mask,sizei,hipMemcpyHostToDevice);
    dim3 dimGrid(ceil(nptsx/(float)TX),ceil(nptsy/(float)TY),1);
    dim3 dimBlock(TX,TY,1);
    for(k=0;k<nsteps;++k){
        performUpdatesKernel<<<dimGrid,dimBlock>>>(d_phi,d_oldphi,d_mask,nptsx,nptsy);
        doCopyKernel<<<dimGrid,dimBlock>>>(d_phi,d_oldphi,d_mask,nptsx,nptsy);
    } 
    hipMemcpy(h_phi,d_oldphi,sizef,hipMemcpyDeviceToHost);
    hipFree(d_phi); hipFree(d_oldphi); hipFree(d_mask);
}
   
int RGBval(float x){
    int R, B, G, pow8 = 256;
    if(x<=0.5){
        B = (int)((1.0-2.0*x)*255.0);
        G = (int)(2.0*x*255.0);
	R = 0; 
    }
    else{
        B = 0;
        G = (int)((2.0-2.0*x)*255.0);
        R = (int)((2.0*x-1.0)*255.0);
    }
    return (B+(G+R*pow8)*pow8);
}

int setup_grid (float  *h_phi, int nptsx, int nptsy, int  *h_mask)
{
    int i, j, nx2, ny2;

    for(j=0;j<nptsy;j++)
       for(i=0;i<nptsx;i++){
          h_phi[j*nptsx+i]  = 0.0;
          h_mask[j*nptsx+i] = 1;
       }

    for(i=0;i<nptsx;i++) h_mask[i] = 0;

    for(i=0;i<nptsx;i++) h_mask[(nptsy-1)*nptsx+i] = 0;

    for(j=0;j<nptsy;j++) h_mask[j*nptsx] = 0;

    for(j=0;j<nptsy;j++) h_mask[j*nptsx+nptsx-1] = 0;

    nx2 = nptsx/2;
    ny2 = nptsy/2;
    h_mask[ny2*nptsx+nx2] = 0;
    h_mask[ny2*nptsx+nx2-1] = 0;
    h_mask[(ny2-1)*nptsx+nx2] = 0;
    h_mask[(ny2-1)*nptsx+nx2-1] = 0;
    h_phi[ny2*nptsx+nx2]  = 1.0;
    h_phi[ny2*nptsx+nx2-1]  = 1.0;
    h_phi[(ny2-1)*nptsx+nx2]  = 1.0;
    h_phi[(ny2-1)*nptsx+nx2-1]  = 1.0;
    return 0;
}

int output_array (float *h_phi, int nptsx, int nptsy)
{
   int i, j, k=0;
   FILE *fp;

   
   fp = fopen("outCUDA.ps","w");
   fprintf(fp,"/picstr %d string def\n",nptsx);
   fprintf(fp,"50 50 translate\n");
   fprintf(fp,"%d %d scale\n",nptsx, nptsy);
   fprintf(fp,"%d %d 8 [%d 0 0 %d 0 %d] \n",nptsx, nptsy, nptsx, nptsy, -nptsx);
   fprintf(fp,"{currentfile 3 200 mul string readhexstring pop} bind false 3 colorimage\n");

   for(j=0;j<nptsy;j++){
        for(i=0;i<nptsx;i++,k++){
             fprintf(fp,"%06x",RGBval(h_phi[j*nptsx+i]));
             if((k+1)%10==0) fprintf(fp,"\n");
        }
   }
   fclose(fp);
   return 0;
}

int main (int argc, char *argv[])
{
   float *h_phi;
   float *h_oldphi;
   int *h_mask;
   int nsize1=sizeof(float)*NPTSX*NPTSY;
   int nsize2=sizeof(int)*NPTSX*NPTSY;

   h_phi = (float *)malloc(nsize1);
   h_oldphi = (float *)malloc(nsize1);
   h_mask = (int *)malloc(nsize2);
   setup_grid (h_oldphi, NPTSX, NPTSY, h_mask);
   performUpdates(h_phi,h_oldphi,h_mask,NPTSX,NPTSY,NSTEPS);
 
   output_array (h_phi, NPTSX, NPTSY);
 
   return 0;
}
